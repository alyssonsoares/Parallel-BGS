#include "hip/hip_runtime.h"
#include "BGS.h"
__global__
void BGS_d(uchar* buffer, int buffer_size, uchar* frame, int frame_size, uchar* frameOut){
	const int x =  blockIdx.x * blockDim.x + threadIdx.x;
    int avg = 0;
    int sum = 0;
    uchar* aux;
    if(x < frame_size){
    	for(int i = 0; i < buffer_size; i++){
 	    aux = buffer + i*frame_size;
            sum += aux[x];
     	}
     	avg = sum / buffer_size;
     	frameOut[x] = frame[x] - avg;
        //calcula a diferença e escreve no frame do vetor resposta
    }
}

void BGS(uchar* buffer, int buffer_size, uchar* frame, int frame_size, uchar* frameOut){
	const dim3 block(BLOCK_SIZE_X,1,1);
	const dim3 grid(frame_size/BLOCK_SIZE_X,1);

	BGS_d<<<block,grid>>>(buffer, buffer_size, frame, frame_size, frameOut)
}

