#include "hip/hip_runtime.h"
#include "MemHandler.h"

#define BLOCK_SIZE_X 8
#define BLOCK_SIZE_Y 8

__global__
void d_putInBuffer(unsigned char* BUFF, const unsigned char* frameAtual, int cols, int rows, int sizeBUFF, int pos){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	int p = i*cols+j;

	BUFF[pos*cols*rows+p] = *(frameAtual+p);
}


void putInBuffer(unsigned char* BUFF, const unsigned char* frameAtual, int cols, int rows, int sizeBUFF, int pos){
	const dim3 block(BLOCK_SIZE_X,BLOCK_SIZE_Y,1);
	const dim3 grid(rows/BLOCK_SIZE_X,cols/BLOCK_SIZE_Y,1);
	
	d_putInBuffer<<<block, grid>>>(BUFF, frameAtual, cols, rows, sizeBUFF, pos);
}


void alloc(unsigned char* &frameEntrada, unsigned char* &frameIntermediario, unsigned char* &frameTratado, unsigned char* &fore, unsigned char* &BUFF, int cols, int rows, int sizeBUFF){

	hipMalloc(&frameEntrada, (cols*rows*sizeof(unsigned char)));
	hipMalloc(&frameIntermediario, cols*rows*sizeof(unsigned char));
	hipMalloc(&frameTratado, cols*rows*sizeof(unsigned char));
	hipMalloc(&fore, cols*rows*sizeof(unsigned char));

	hipMalloc(&BUFF, cols*rows*sizeof(unsigned char)*sizeBUFF);



}
void dealloc(unsigned char* &frameEntrada, unsigned char* &frameIntermediario, unsigned char* &frameTratado, unsigned char* &fore, unsigned char* &BUFF, int cols, int rows, int sizeBUFF){

	hipFree(frameEntrada);
	hipFree(frameIntermediario);
	hipFree(frameTratado);
	hipFree(fore);

	hipFree(BUFF);

}

