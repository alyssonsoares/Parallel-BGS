#include "hip/hip_runtime.h"
#include "Filtros.h"
__global__
void gaussian_blur_d(const unsigned char* const inputChannel,
                   unsigned char* const outputChannel,
                   int numRows, int numCols,
                   const float* const filter, const int filterWidth)
{

    const int x =  blockIdx.x * blockDim.x + threadIdx.x;
    const int y =  blockIdx.y * blockDim.y + threadIdx.y;
    const int m = y * numCols + x;
    
    if(x >= numCols || y >= numRows)
         return;
    
    float color = 0.0f;
    
    for(int f_y = 0; f_y < filterWidth; f_y++) {
        for(int f_x = 0; f_x < filterWidth; f_x++) {
   
            int c_x = x + f_x - filterWidth/2;
            int c_y = y + f_y - filterWidth/2;
            c_x = min(max(c_x, 0), numCols - 1);
            c_y = min(max(c_y, 0), numRows - 1);
            float filter_value = filter[f_y*filterWidth + f_x];
            color += filter_value*static_cast<float>(inputChannel[c_y*numCols + c_x]);
            
        }
    }
    
    outputChannel[m] = color;
  
}


__global__
void rgb_to_greyscale_d(const unsigned char* const bgrImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{

 int i = blockIdx.x * blockDim.x + threadIdx.x;// determina iteração das linhas
 int j = blockIdx.y * blockDim.y + threadIdx.y; //determina iteração das colunas
    if(i<numRows && j< numCols){
       //Y = 0.2126R + 0.7152G + 0.0722B
       float b = *(bgrImage+i*numCols+j);
       float g = *(bgrImage+1*numCols*numRows+i*numCols+j);
       float r = *(bgrImage+2*numCols*numRows+i*numCols+j);
       *(greyImage+i*numCols+j) = (unsigned char)(0.2126*r + 0.7152*g + 0.722*b);
    }
     
}

void gaussian_blur(const unsigned char* const inputChannel,
                   unsigned char* const outputChannel,
                   int numRows, int numCols,
                   const float* const filter, const int filterWidth){

	const dim3 block(BLOCK_SIZE_X,BLOCK_SIZE_Y,1);
	const dim3 grid(rows/BLOCK_SIZE_X,cols/BLOCK_SIZE_Y,1);
	gaussian_blur_d<<<block,grid>>>(inputChannel, outputChannel, numRows, numCols, filter, filterWidth);
}


void rgb_to_greyscale(const unsigned char* const bgrImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols);
	const dim3 block(BLOCK_SIZE_X,BLOCK_SIZE_Y,1);
	const dim3 grid(rows/BLOCK_SIZE_X,cols/BLOCK_SIZE_Y,1);
	rgb_to_greyscale_d<<<block,grid>>>(bgrImage, greyImage, numRows, numCols);

